#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <assert.h>

#include "matrix.h"
#include "strassen.h"
#include "timer.hpp"

int main(int argc, char** argv) {
	if (argc != 4) {
		fprintf(stderr, "Usage: %s <matrices size> <threshold> <cublas>\n", argv[0]);
		fputs("cublas:\t0: use Strassen with naive algorithm with SM\n\t1: use Strassen with cuBLAS' algorithm\n", stderr);
		exit(0);
	}

	int n = atoi(argv[1]);
	int threshold = atoi(argv[2]);
	cublas = (bool) atoi(argv[3]);

	if (!isPowerOfTwo(n)) {
		/* rectangular matrices */
		fputs("WARNING: matrices size is not power of two\n", stdout);
		n = pow(2, ceil(log2((float) n)));
		printf("dimension of matrices = %d\n", n);
	}
	
	dim3 blocks((n + BLOCK_DIM - 1) / BLOCK_DIM, (n + BLOCK_DIM - 1) / BLOCK_DIM);
	dim3 threads(BLOCK_DIM, BLOCK_DIM);

	if (n < threshold && threshold < BLOCK_DIM) {
		fputs("ERROR: matrix size is less than threshold\n", stderr);
		exit(-2);
	}

	float *A_host = (float *) calloc(n * n, sizeof(float));
	float *B_host = (float *) calloc(n * n, sizeof(float));
	float *C_host = (float *) calloc(n * n, sizeof(float));
	float *tmp = (float *) calloc(n * n, sizeof(float));

	srand(time(NULL));
	initm(A_host, n, n);
	initm(B_host, n, n);

	printm(A_host, n, n, "A");
	printm(B_host, n, n, "B");

	/* local matrices */
	int depth, dim = n;
	for (depth = 0; depth < DEPTH && dim > 0; ++depth, dim /= 2) {
		hipMalloc((float **) &A_dev[depth], dim * dim * sizeof(float));
		hipMalloc((float **) &B_dev[depth], dim * dim * sizeof(float));

		if (depth == 0)
			hipMalloc((float **) &C_dev, dim * dim * sizeof(float));
		else {
			hipMalloc((float **) &M1[depth], dim * dim * sizeof(float));
			hipMalloc((float **) &M2[depth], dim * dim * sizeof(float));
			hipMalloc((float **) &M3[depth], dim * dim * sizeof(float));
			hipMalloc((float **) &M4[depth], dim * dim * sizeof(float));
			hipMalloc((float **) &M5[depth], dim * dim * sizeof(float));
			hipMalloc((float **) &M6[depth], dim * dim * sizeof(float));
			hipMalloc((float **) &M7[depth], dim * dim * sizeof(float));
		}
	}

	hipMemcpy(A_dev[0], A_host, n * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_dev[0], B_host, n * n * sizeof(float), hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	Timer t;

	t.start();
	matmul<<<blocks, threads>>>(C_dev, A_dev[0], B_dev[0], n);
	hipDeviceSynchronize();
	t.stop();

	printf("time for naive: %.5fms\n", t.get());
	hipMemcpy(tmp, C_dev, n * n * sizeof(float), hipMemcpyDeviceToHost);

	t.start();
	strassen(handle, C_dev, A_dev[0], B_dev[0], n, 0, threshold);
	t.stop();

	printf("time for strassen: %.5fms\n", t.get());
	hipMemcpy(C_host, C_dev, n * n * sizeof(float), hipMemcpyDeviceToHost);

	if (cublas) {
		t.start();
		cublasMatmul(handle, C_dev, A_dev[0], B_dev[0], n);
		t.stop();
		
		printf("time for cublas: %.5fms\n", t.get());
		hipMemcpy(C_host, C_dev, n * n * sizeof(float), hipMemcpyDeviceToHost);
	}

	hipblasDestroy(handle);

	int i;
	for (i = 0; i < depth; ++i) {
		hipFree(A_dev[i]); hipFree(B_dev[i]);

		if (i == 0)
			hipFree(C_dev);
		else {
			hipFree(M1[i]);
			hipFree(M2[i]);
			hipFree(M3[i]);
			hipFree(M4[i]);
			hipFree(M5[i]);
			hipFree(M6[i]);
			hipFree(M7[i]);
		}
	}

	printm(C_host, n, n, "C");
	printf(matcmp(tmp, C_host, n) ? "Strassen OK\n" : "Strassen not working\n");

	hipDeviceReset();

	free(A_host); free(B_host); free(C_host); free(tmp);

	return 0;
}